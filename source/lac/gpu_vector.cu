#include "hip/hip_runtime.h"
#include <deal.II/lac/gpu_vector.h>
#include <cstdio>

DEAL_II_NAMESPACE_OPEN

__global__ void foo_kernel(float *x, int N)
{
  const unsigned int idx = threadIdx.x + blockIdx.x*blockDim.x;
  if(idx<N)
    x[idx] = 1.3;
}


GpuVector::GpuVector(int N)
{
  this->N=N;
  hipMalloc(&x,sizeof(float)*N);
}

GpuVector::~GpuVector()
{
  hipFree(x);
}

void GpuVector::foo()
{
  dim3 bk_dim(128);
  dim3 gd_dim(1+(N-1)/128);
  foo_kernel<<<gd_dim,bk_dim>>>(this->x,this->N);
}

void GpuVector::hello()
{
  float x_host;
  hipMemcpy(&x_host,x,sizeof(float),hipMemcpyDeviceToHost);
  printf("x[0]=%g\n",x_host);
}

DEAL_II_NAMESPACE_CLOSE
